#include "hip/hip_runtime.h"
/*
* Copyright 2010-2014 METU, Middle East Technical University, Informatics Institute
*
* This file is part of "Unwrapping OmniDirectional Images" Library.
*
* "Unwrapping OmniDirectional Images" is free software: 
* you can redistribute it and/or modify
* it under the terms of the GNU General Public License as published by
* the Free Software Foundation, either version 3 of the License, or
* (at your option) any later version.
*
* "Unwrapping OmniDirectional Images" is distributed in the hope 
* that it will be useful, but WITHOUT ANY WARRANTY; without 
* even the implied warranty of MERCHANTABILITY or FITNESS FOR A 
* PARTICULAR PURPOSE.  See the GNU General Public License for more details.
*
* You should have received a copy of the GNU General Public License.
* If not, see <http://www.gnu.org/licenses/>. 
* Please contact Alptekin TEMIZEL for more info about 
* licensing atemizel@metu.edu.tr.
*
*/

/*
* Unwrapper.cu
* 
* Author: Ilker GURCAN
*/

#include<cstring>
#include<math.h>
#include<opencv2\core\core.hpp>
#include "Unwrapper.cuh"

__host__
Unwrapper::Unwrapper(int resWidth, 
					 int resHeight, 
					 float xCenter, 
					 float yCenter, 
					 float radius, 
					 float alfa, 
					 float beta) {

	this->resWidth  = resWidth;
	this->resHeight = resHeight;
	this->xCenter   = xCenter;
	this->yCenter   = yCenter;
	this->radius    = radius;
	this->alfa      = alfa;
	this->beta      = beta;
	texture         = 0;
	cuArray         = 0;
	dUnwrappedImg   = NULL;
}

__host__
void Unwrapper::init(int imW, int imH, size_t stride, int numChannels) {

	this->imW         = imW;
	this->imH         = imH;
	this->stride      = stride;
	this->numChannels = numChannels;
	initTexture();
	//Allocate Data for Output...
	hipMalloc(&dUnwrappedImg, resWidth * resHeight * sizeof(unsigned char) * numChannels);
}

__host__
void Unwrapper::initTexture() {

	struct hipResourceDesc resDesc;
	struct hipTextureDesc texDesc;
	struct hipChannelFormatDesc channelDesc;

	if(numChannels == 4)
		channelDesc = hipCreateChannelDesc(8, 8, 8, 8, hipChannelFormatKindUnsigned);
	else 
		channelDesc = hipCreateChannelDesc(8, 0, 0, 0, hipChannelFormatKindUnsigned);
	//Allocate Texture Data on GPU...
	hipMallocArray(&cuArray, &channelDesc, stride / numChannels, imH);
	//Specify texture Data...
	memset(&resDesc, 0, sizeof(resDesc));
	resDesc.resType         = hipResourceTypeArray;
	resDesc.res.array.array = cuArray;
	//Specify texture Object Parameters...
	memset(&texDesc, 0, sizeof(texDesc));
	texDesc.addressMode[0]   = hipAddressModeWrap;
	texDesc.addressMode[1]   = hipAddressModeWrap;
	texDesc.filterMode       = hipFilterModePoint;
	texDesc.normalizedCoords = 0;
	texDesc.readMode         = hipReadModeElementType;
	//Create Texture Object...
	hipCreateTextureObject(&texture, &resDesc, &texDesc, NULL);
}

__host__
Unwrapper::~Unwrapper() {

	if(texture)
		hipDestroyTextureObject(texture);
	if(cuArray)
		hipFreeArray(cuArray);
	if(dUnwrappedImg != NULL)
		hipFree(dUnwrappedImg);
}

/**
* !!!!!!!!!!!!!!!!!!!!!!Texture Objects Require Compute Capability >= 3.0!!!!!!!!!!!!!!!!!!!!!
**/
template <typename T>
__global__ void unwrapImageKernel(hipTextureObject_t texture, 
								  T *dUnwrappedImg, 
								  float radius, 
								  float xCenter, 
								  float yCenter, 
								  int width, 
								  int height, 
								  float oneOverAlfa, 
								  float oneOverBeta) { 
	
	const int bId      = blockIdx.x;
	const int tGId     = bId * MAX_BLOCK_SIZE + threadIdx.x;
	const int tWId     = tGId & (WARP_SIZE - 1);
	const int warpGId  = tGId / WARP_SIZE;
	const int gridSize = WARPS_PER_BLOCK * WARP_SIZE * gridDim.x;
	//Variables Used Within The Algorithm...
	float sourceX, sourceY;

	for(int row = warpGId; row < height; row+=gridSize) { 
		for(int i = tWId; i < width; i+=WARP_SIZE) { 
			//Scan The Oval Image In The Reverse Direction (That is Why We Multiply by -1)...
			sourceX = (xCenter + oneOverAlfa * row * __cosf(-1 * i * oneOverBeta / radius));
			sourceY = (yCenter + oneOverAlfa * row * __sinf(-1 * i * oneOverBeta / radius));
			//Origin Of The Coordinate System For The Destination Image Is Bottom-Left Corner Of It...
			//In Other Words--->(0,height)====>Origin Of The Coordinate System...
			const int curIndex = width * ((height - 1) - row) + i;
			//Nearest Neighbour...
			dUnwrappedImg[curIndex] = tex2D<T>(texture, sourceX, sourceY);
		}//End Of Inner-for Loop...
	}//End Of Outer-for Loop...
	
}//End Of flatImageKernel Method...

unsigned char* Unwrapper::unwrap(unsigned char *inputImg) {

	int numWarps  = resHeight;
	int numBlocks = DIVIDE_INTO(WARP_SIZE * numWarps, MAX_BLOCK_SIZE);
	dim3 gridDim(numBlocks, 1, 1);
	dim3 blockDim(MAX_BLOCK_SIZE, 1, 1);

	//Move Data to GPU...
	hipMemcpyToArray(cuArray, 
		              0, 
					  0, 
					  (void*)inputImg, 
					  stride * imH, 
					  hipMemcpyHostToDevice);
	if(numChannels == 4) 
		unwrapImageKernel<uchar4><<<gridDim, blockDim>>>(texture, 
													     reinterpret_cast<uchar4*>(dUnwrappedImg), 
														 radius, 
														 xCenter, 
														 yCenter, 
														 resWidth, 
														 resHeight, 
														 1.0f / alfa, 
														 1.0f / beta);
	else 
		unwrapImageKernel<uchar><<<gridDim, blockDim>>>(texture, 
														reinterpret_cast<uchar*>(dUnwrappedImg), 
														radius, 
														xCenter, 
														yCenter, 
														resWidth, 
														resHeight, 
														1.0f / alfa, 
														1.0f / beta);
	hipDeviceSynchronize();
	return reinterpret_cast<unsigned char*>(dUnwrappedImg);
}
